#include "hip/hip_runtime.h"
#include "handler.h"

template<class T>
__global__ void dev_add(T* a, T* b, T* res, int N) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  while (idx < N) {
    res[idx] = a[idx] + b[idx];
    idx += blockDim.x * gridDim.x;
  }
};

template<class T>
void add(T* a, T* b, T* res, int N) {
  T *dev_a, *dev_b, *dev_res;
  
  HANDLE_ERROR( hipMalloc((void**)&dev_a, N * sizeof(double)) );
  HANDLE_ERROR( hipMalloc((void**)&dev_b, N * sizeof(double)) );
  HANDLE_ERROR( hipMalloc((void**)&dev_res, N * sizeof(double)) );

  HANDLE_ERROR( hipMemcpy(dev_a, a, N * sizeof(double), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(dev_b, b, N * sizeof(double), hipMemcpyHostToDevice) );

  HANDLE_ERROR( dev_add<<<128,128>>>(dev_a, dev_b, dev_res, N) );

  HANDLE_ERROR( hipMemcpy(res, dev_res, N * sizeof(double), hipMemcpyDeviceToHost) );

  hipFree(dev_a); hipFree(dev_b); hipFree(dev_res);
}

void util() {
  double* a;
  add(a,a,a,1);
  
  long* b;
  add(b,b,b,1);
}
