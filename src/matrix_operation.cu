#include "hip/hip_runtime.h"
#include "handler.h"

template<class T>
__global__ void dev_add(T* a, T* b, T* res, int N) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  while (idx < N) {
    res[idx] = a[idx] + b[idx];
    idx += blockDim.x * gridDim.x;
  }
};

template<class T>
void cuda_add(T* a, T* b, T* res, int N) {
  T *dev_a, *dev_b, *dev_res;
  
  HANDLE_ERROR( hipMalloc((void**)&dev_a, N * sizeof(T)) );
  HANDLE_ERROR( hipMalloc((void**)&dev_b, N * sizeof(T)) );
  HANDLE_ERROR( hipMalloc((void**)&dev_res, N * sizeof(T)) );

  HANDLE_ERROR( hipMemcpy(dev_a, a, N * sizeof(T), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(dev_b, b, N * sizeof(T), hipMemcpyHostToDevice) );

  dev_add<<<128,128>>>(dev_a, dev_b, dev_res, N);

  HANDLE_ERROR( hipMemcpy(res, dev_res, N * sizeof(T), hipMemcpyDeviceToHost) );

  hipFree(dev_a); hipFree(dev_b); hipFree(dev_res);
}


template<class T>
__global__ void dev_transpose(T* input, T* res, int M, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    while(idx < M*N) {
      int i = idx % M;
      int j = idx / N;

      res[N*i + j] = input[M*j + i];

      idx += blockDim.x * gridDim.x;
    }
};

template<class T>
void cuda_transpose(T* input, T* res, int M, int N) {
    T *dev_input, *dev_result;

    HANDLE_ERROR( hipMalloc((void**)&dev_input, M * N * sizeof(T)) );
    HANDLE_ERROR( hipMalloc((void**)&dev_result, M * N * sizeof(T)) );

    HANDLE_ERROR( hipMemcpy(dev_input, input, M * N * sizeof(T), hipMemcpyHostToDevice) );

    dev_transpose<<<128,128>>>(dev_input, dev_result, M, N);

    HANDLE_ERROR( hipMemcpy(res, dev_result, M * N * sizeof(T), hipMemcpyDeviceToHost) );

    hipFree(dev_input); hipFree(dev_result);
};



void util() {
  // this function is needed so that the compiler compiles those functions above in the object file

  double* a;
  cuda_add(a,a,a,1);
  cuda_transpose(a,a,1,1);
  
  long* b;
  cuda_add(b,b,b,1);
  cuda_transpose(b,b,1,1);
}
