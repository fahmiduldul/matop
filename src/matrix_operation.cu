
#include <hip/hip_runtime.h>
template<class T>
__global__ void dev_add(T* a, T* b, T* res, int N) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  while (idx < N) {
    res[idx] = a[idx] + b[idx];
    idx += blockDim.x * gridDim.x;
  }
};

template<class T>
void add(T* a, T* b, T* res, int N) {
  T *dev_a, *dev_b, *dev_res;
  
  hipMalloc((void**)&dev_a, N * sizeof(double));
  hipMalloc((void**)&dev_b, N * sizeof(double));
  hipMalloc((void**)&dev_res, N * sizeof(double));

  hipMemcpy(dev_a, a, N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N * sizeof(double), hipMemcpyHostToDevice);

  dev_add<<<128,128>>>(dev_a, dev_b, dev_res, N);

  hipMemcpy(res, dev_res, N * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(dev_a); hipFree(dev_b); hipFree(dev_res);
}